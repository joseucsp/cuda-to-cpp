#include "hip/hip_runtime.h"
#include "func.h"
#include "cudaHeaders.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[idx]  = a[idx] + b[idx];
}

void VectorAdd(int *C, const int *A, const int *B)
{
    int *devA = new int[5];
    int *devB = new int[5];
    int *devC = new int[5];

    hipMalloc((void**)&devA, sizeof(int)*5);
    hipMemcpy(devA, A, sizeof(int)*5, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&devB, sizeof(int)*5);
    hipMemcpy(devB, B, sizeof(int)*5, hipMemcpyHostToDevice);   
 
    hipMalloc((void**)&devC, sizeof(int)*5);
    hipMemcpy(devC, C, sizeof(int)*5, hipMemcpyHostToDevice);
    
    addKernel<<<1,32>>>(devC, devA, devB);    
    
    hipMemcpy(C, devC, sizeof(int)*5, hipMemcpyDeviceToHost);

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);    
}